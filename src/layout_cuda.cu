#include "hip/hip_runtime.h"
/*
 * This file is part of HGVisualizer.
 *
 * Copyright (C) Joe Mruz
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */
#include "layout_cuda.h"

using namespace layout_cuda;

namespace {
// CUDA kernel for O(n^2) force-directed layout (repulsion + attraction)
__global__ void kernel_compute_forces(
    const CudaVec2* positions,
    CudaVec2* forces,
    int n,
    float repulsion_constant,
    const CudaEdge* edges,
    int edge_count,
    float spring_length,
    float attraction_constant) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    CudaVec2 force(0, 0);
    CudaVec2 vi = positions[i];

    // Repulsion
    for (int j = 0; j < n; ++j) {
        if (i == j) continue;
        CudaVec2 vj = positions[j];
        CudaVec2 delta = vi - vj;

        // Note: When two nodes (or a node and a center of mass) are extremely close, 
        // the repulsive force (which is proportional to 1/(dist*dist)) becomes 
        // enormous, causing a node to "explode" away. Clamp the minimum distance to 
        // a reasonable value proportional to the graph's scale. Or we can add a softening factor.
        float dist = fmaxf(delta.length(), 20.0f); // Clamp to minimum distance -- tune as needed
        force += delta.normalized() * (repulsion_constant / (dist * dist));
    }

    // Attraction (springs)
    for (int e = 0; e < edge_count; ++e) {
        if (edges[e].from == i || edges[e].to == i) {
            int other = (edges[e].from == i) ? edges[e].to : edges[e].from;
            CudaVec2 vj = positions[other];
            CudaVec2 delta = vj - vi;
            float dist = fmaxf(delta.length(), 1.0f); // Clamp to minimum distance
            force += delta.normalized() * (dist - spring_length) * attraction_constant;
        }
    }

    forces[i] = force;

    // Cap the maximum force to prevent exploding nodes
    float max_force = 1000.0f; // Tune as needed
    float force_len = force.length();
    if (force_len > max_force) {
        forces[i] = force.normalized() * max_force;
    } else {
        forces[i] = force;
    }
}
}

// Host-side wrapper
void layout_cuda::compute_forces(
    const CudaVec2* h_positions,
    CudaVec2* h_forces,
    int n,
    float repulsion_constant,
    const CudaEdge* h_edges,
    int edge_count,
    float spring_length,
    float attraction_constant) {
        
    CudaVec2* d_positions = nullptr;
    CudaVec2* d_forces = nullptr;
    CudaEdge* d_edges = nullptr;
    hipMalloc(&d_positions, n * sizeof(CudaVec2));
    hipMalloc(&d_forces, n * sizeof(CudaVec2));
    hipMalloc(&d_edges, edge_count * sizeof(CudaEdge));
    hipMemcpy(d_positions, h_positions, n * sizeof(CudaVec2), hipMemcpyHostToDevice);
    hipMemcpy(d_edges, h_edges, edge_count * sizeof(CudaEdge), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    kernel_compute_forces<<<gridSize, blockSize>>>(
        d_positions, d_forces, n, repulsion_constant,
        d_edges, edge_count, spring_length, attraction_constant
    );

    hipMemcpy(h_forces, d_forces, n * sizeof(CudaVec2), hipMemcpyDeviceToHost);
    hipFree(d_positions);
    hipFree(d_forces);
    hipFree(d_edges);
}